#include "hip/hip_runtime.h"
#include "foo.h"
#include <hip/hip_runtime_api.h>

__global__ void k_foo(int32_t* val)
{
	*val += 1;
}

int32_t foo(int32_t input)
{
	int32_t* buf;
	hipMalloc((void**)&buf, sizeof(int32_t));
	hipMemcpy(buf, &input, sizeof(int32_t), hipMemcpyHostToDevice);
	k_foo<<<1,1>>>(buf);
	hipMemcpy(&input, buf, sizeof(int32_t), hipMemcpyDeviceToHost);
	hipFree(buf);
	return input;
}
